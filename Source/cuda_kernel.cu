#include "hip/hip_runtime.h"
#include "SparseFrame.h"

__global__ void mappedSubtract_kernel ( Float *d_A, Long nsrow, Float *d_C, Long nccol, Long ncrow, Long *d_RelativeMap )
{
    Long cj, ci;

    cj = blockIdx.x * blockDim.x + threadIdx.x;
    ci = blockIdx.y * blockDim.y + threadIdx.y;

    if ( cj < nccol && ci < ncrow )
        d_A [ d_RelativeMap[cj] * nsrow + d_RelativeMap[ci] ] -= d_C [ cj * ncrow + ci ];
}

void mappedSubtract ( Float *d_A, Long nsrow, Float *d_C, Long nccol, Long ncrow, Long *d_RelativeMap, hipStream_t stream )
{
    dim3 block;
    dim3 thread(16, 16);

    block.x = ( nccol + thread.x - 1 ) / thread.x;
    block.y = ( ncrow + thread.y - 1 ) / thread.y;

    mappedSubtract_kernel <<< block, thread, 0, stream >>> ( d_A, nsrow, d_C, nccol, ncrow, d_RelativeMap );
}

__global__ void mappedSubtractComplex_kernel ( Complex *d_A, Long nsrow, Complex *d_C, Long nccol, Long ncrow, Long *d_RelativeMap )
{
    Long cj, ci;

    cj = blockIdx.x * blockDim.x + threadIdx.x;
    ci = blockIdx.y * blockDim.y + threadIdx.y;

    if ( cj < nccol && ci < ncrow )
    {
        d_A [ d_RelativeMap[cj] * nsrow + d_RelativeMap[ci] ].x -= d_C [ cj * ncrow + ci ].x;
        d_A [ d_RelativeMap[cj] * nsrow + d_RelativeMap[ci] ].y -= d_C [ cj * ncrow + ci ].y;
    }
}

void mappedSubtractComplex ( Complex *d_A, Long nsrow, Complex *d_C, Long nccol, Long ncrow, Long *d_RelativeMap, hipStream_t stream )
{
    dim3 block;
    dim3 thread(16, 16);

    block.x = ( nccol + thread.x - 1 ) / thread.x;
    block.y = ( ncrow + thread.y - 1 ) / thread.y;

    mappedSubtractComplex_kernel <<< block, thread, 0, stream >>> ( d_A, nsrow, d_C, nccol, ncrow, d_RelativeMap );
}

#include "hip/hip_runtime.h"
#include "SparseFrame.h"

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
        (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val +
                    __longlong_as_double(assumed)));

    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

__global__ void createRelativeMap_kernel ( Long *d_RelativeMap, Long di_offset, Long *d_Map, Long *d_Lsi, Long dip_offset, Long ldd )
{
    Long di;

    di = di_offset + blockIdx.x * blockDim.x + threadIdx.x;

    if ( di < ldd )
        d_RelativeMap[di] = d_Map [ d_Lsi [ dip_offset + di ] ];
}

void createRelativeMap ( Long *d_RelativeMap, Long di_offset, Long *d_Map, Long *d_Lsi, Long dip_offset, Long ldd, hipStream_t stream )
{
    dim3 block, thread;

    thread.x = CUDA_BLOCKDIM_X * CUDA_BLOCKDIM_Y;
    block.x = ( ldd + thread.x - 1 ) / thread.x;

    createRelativeMap_kernel <<< block, thread, 0, stream >>> ( d_RelativeMap, di_offset, d_Map, d_Lsi, dip_offset, ldd );
}

__global__ void createRelativeMap_batched_kernel ( Long **d_RelativeMap, Long *di_offset, Long **d_Map, Long *d_Lsi, Long *dip_offset, Long *ldd )
{
    Long di;

    di = di_offset[blockIdx.x] + threadIdx.x;

    if ( di < ldd[blockIdx.x] )
        d_RelativeMap[blockIdx.x][di] = d_Map [blockIdx.x] [ d_Lsi [ dip_offset[blockIdx.x] + di ] ];
}

void createRelativeMap_batched ( Long batchSize, Long **d_RelativeMap, Long *di_offset, Long **d_Map, Long *d_Lsi, Long *dip_offset, Long *ldd, hipStream_t stream )
{
    dim3 block, thread;

    thread.x = CUDA_BLOCKDIM_X * CUDA_BLOCKDIM_Y;
    block.x = batchSize;

    createRelativeMap_batched_kernel <<< block, thread, 0, stream >>> ( d_RelativeMap, di_offset, d_Map, d_Lsi, dip_offset, ldd );
}

__global__ void mappedSubtract_kernel ( int isAtomic, int isComplex, void *d_A, Long lda, void *d_C, Long cj_offset, Long ci_offset, Long nccol, Long ncrow, Long ldc, Long *d_RelativeMap )
{
    Long cj, ci;

    cj = cj_offset + blockIdx.x * blockDim.x + threadIdx.x;
    ci = ci_offset + blockIdx.y * blockDim.y + threadIdx.y;

    if ( !isAtomic )
    {
        if ( !isComplex )
        {
            if ( cj < nccol && ci < ncrow )
                ( (Float*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ] -= ( (Float*) d_C )  [ cj * ldc + ci ];
        }
        else
        {
            if ( cj < nccol && ci < ncrow )
            {
                ( (Complex*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ].x -= ( (Complex*) d_C ) [ cj * ldc + ci ].x;
                ( (Complex*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ].y -= ( (Complex*) d_C ) [ cj * ldc + ci ].y;
            }
        }
    }
    else
    {
        if ( !isComplex )
        {
            if ( cj < nccol && ci < ncrow )
                atomicAdd ( & ( ( (Float*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ] ), - ( (Float*) d_C ) [ cj * ldc + ci ] );
        }
        else
        {
            if ( cj < nccol && ci < ncrow )
            {
                atomicAdd ( & ( ( (Complex*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ].x ), - ( (Complex*) d_C ) [ cj * ldc + ci ].x );
                atomicAdd ( & ( ( (Complex*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ].y ), - ( (Complex*) d_C ) [ cj * ldc + ci ].y );
            }
        }
    }
}

void mappedSubtract ( int isAtomic, int isComplex, void *d_A, Long lda, void *d_C, Long cj_offset, Long ci_offset, Long nccol, Long ncrow, Long ldc, Long *d_RelativeMap, hipStream_t stream )
{
    dim3 block, thread;

    thread.x = CUDA_BLOCKDIM_X;
    thread.y = CUDA_BLOCKDIM_Y;
    block.x = ( nccol + thread.x - 1 ) / thread.x;
    block.y = ( ncrow + thread.y - 1 ) / thread.y;

    mappedSubtract_kernel <<< block, thread, 0, stream >>> ( isAtomic, isComplex, d_A, lda, d_C, cj_offset, ci_offset, nccol, ncrow, ldc, d_RelativeMap );
}

__global__ void mappedSubtract_batched_kernel ( int isAtomic, int isComplex, void **d_A, Long *lda, void **d_C, Long *cj_offset, Long *ci_offset, Long *nccol, Long *ncrow, Long *ldc, Long **d_RelativeMap )
{
    Long cj, ci;

    cj = cj_offset[blockIdx.x] + threadIdx.x;
    ci = ci_offset[blockIdx.x] + threadIdx.y;

    if ( !isAtomic )
    {
        if ( !isComplex )
        {
            if ( cj < nccol[blockIdx.x] && ci < ncrow[blockIdx.x] )
                ( (Float*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ] -= ( (Float*) ( d_C[blockIdx.x] ) )  [ cj * ldc[blockIdx.x] + ci ];
        }
        else
        {
            if ( cj < nccol[blockIdx.x] && ci < ncrow[blockIdx.x] )
            {
                ( (Complex*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ].x -= ( (Complex*) ( d_C[blockIdx.x] ) ) [ cj * ldc[blockIdx.x] + ci ].x;
                ( (Complex*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ].y -= ( (Complex*) ( d_C[blockIdx.x] ) ) [ cj * ldc[blockIdx.x] + ci ].y;
            }
        }
    }
    else
    {
        if ( !isComplex )
        {
            if ( cj < nccol[blockIdx.x] && ci < ncrow[blockIdx.x] )
                atomicAdd ( & ( ( (Float*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ] ), - ( (Float*) ( d_C[blockIdx.x] ) ) [ cj * ldc[blockIdx.x] + ci ] );
        }
        else
        {
            if ( cj < nccol[blockIdx.x] && ci < ncrow[blockIdx.x] )
            {
                atomicAdd ( & ( ( (Complex*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ].x ), - ( (Complex*) ( d_C[blockIdx.x] ) ) [ cj * ldc[blockIdx.x] + ci ].x );
                atomicAdd ( & ( ( (Complex*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ].y ), - ( (Complex*) ( d_C[blockIdx.x] ) ) [ cj * ldc[blockIdx.x] + ci ].y );
            }
        }
    }
}

void mappedSubtract_batched ( Long batchSize, int isAtomic, int isComplex, void **d_A, Long *lda, void **d_C, Long *cj_offset, Long *ci_offset, Long *nccol, Long *ncrow, Long *ldc, Long **d_RelativeMap, hipStream_t stream )
{
    dim3 block, thread;

    thread.x = CUDA_BLOCKDIM_X;
    thread.y = CUDA_BLOCKDIM_Y;
    block.x = batchSize;

    mappedSubtract_batched_kernel <<< block, thread, 0, stream >>> ( isAtomic, isComplex, d_A, lda, d_C, cj_offset, ci_offset, nccol, ncrow, ldc, d_RelativeMap );
}

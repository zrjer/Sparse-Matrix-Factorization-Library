#include "hip/hip_runtime.h"
#include "SparseFrame.h"

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
        (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val +
                    __longlong_as_double(assumed)));

    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

__global__ void createRelativeMap_kernel ( Long *d_RelativeMap, Long *d_Map, Long *d_Lsi, Long di_offset, Long ldd )
{
    Long di;

    di = blockIdx.x * blockDim.x + threadIdx.x;

    if ( di < ldd )
        d_RelativeMap[di] = d_Map [ d_Lsi [ di_offset + di ] ];
}

void createRelativeMap ( Long *d_RelativeMap, Long *d_Map, Long *d_Lsi, Long di_offset, Long ldd, hipStream_t stream )
{
    dim3 block;
    dim3 thread(256);

    block.x = ( ldd + thread.x - 1 ) / thread.x;

    createRelativeMap_kernel <<< block, thread, 0, stream >>> ( d_RelativeMap, d_Map, d_Lsi, di_offset, ldd );
}

__global__ void createRelativeMap_batched_kernel ( Long **d_RelativeMap, Long **d_Map, Long *d_Lsi, Long *di_offset, Long *ldd )
{
    Long di;

    for ( di = threadIdx.x; di < ldd[blockIdx.x]; di += blockDim.x )
        if ( di < ldd[blockIdx.x] )
            d_RelativeMap[blockIdx.x][di] = d_Map [blockIdx.x] [ d_Lsi [ di_offset[blockIdx.x] + di ] ];
}

void createRelativeMap_batched ( Long batchSize, Long **d_RelativeMap, Long **d_Map, Long *d_Lsi, Long *di_offset, Long *ldd, hipStream_t stream )
{
    dim3 block;
    dim3 thread(256);

    block.x = batchSize;

    createRelativeMap_batched_kernel <<< block, thread, 0, stream >>> ( d_RelativeMap, d_Map, d_Lsi, di_offset, ldd );
}

__global__ void mappedSubtract_kernel ( int isAtomic, int isComplex, void *d_A, Long lda, void *d_C, Long nccol, Long ncrow, Long ldc, Long *d_RelativeMap )
{
    Long cj, ci;

    cj = blockIdx.x * blockDim.x + threadIdx.x;
    ci = blockIdx.y * blockDim.y + threadIdx.y;

    if ( !isAtomic )
    {
        if ( !isComplex )
        {
            if ( cj < nccol && ci < ncrow )
                ( (Float*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ] -= ( (Float*) d_C )  [ cj * ldc + ci ];
        }
        else
        {
            if ( cj < nccol && ci < ncrow )
            {
                ( (Complex*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ].x -= ( (Complex*) d_C ) [ cj * ldc + ci ].x;
                ( (Complex*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ].y -= ( (Complex*) d_C ) [ cj * ldc + ci ].y;
            }
        }
    }
    else
    {
        if ( !isComplex )
        {
            if ( cj < nccol && ci < ncrow )
                atomicAdd ( & ( ( (Float*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ] ), - ( (Float*) d_C ) [ cj * ldc + ci ] );
        }
        else
        {
            if ( cj < nccol && ci < ncrow )
            {
                atomicAdd ( & ( ( (Complex*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ].x ), - ( (Complex*) d_C ) [ cj * ldc + ci ].x );
                atomicAdd ( & ( ( (Complex*) d_A ) [ d_RelativeMap[cj] * lda + d_RelativeMap[ci] ].y ), - ( (Complex*) d_C ) [ cj * ldc + ci ].y );
            }
        }
    }
}

void mappedSubtract ( int isAtomic, int isComplex, void *d_A, Long lda, void *d_C, Long nccol, Long ncrow, Long ldc, Long *d_RelativeMap, hipStream_t stream )
{
    dim3 block;
    dim3 thread(16, 16);

    block.x = ( nccol + thread.x - 1 ) / thread.x;
    block.y = ( ncrow + thread.y - 1 ) / thread.y;

    mappedSubtract_kernel <<< block, thread, 0, stream >>> ( isAtomic, isComplex, d_A, lda, d_C, nccol, ncrow, ldc, d_RelativeMap );
}

__global__ void mappedSubtract_batched_kernel ( int isAtomic, int isComplex, void **d_A, Long *lda, void **d_C, Long *nccol, Long *ncrow, Long *ldc, Long **d_RelativeMap )
{
    Long cj, ci;

    if ( !isAtomic )
    {
        if ( !isComplex )
        {
            for ( cj = threadIdx.x; cj < nccol[blockIdx.x]; cj += blockDim.x )
                for ( ci = threadIdx.y; ci < nccol[blockIdx.x]; ci += blockDim.x )
                    if ( cj < nccol[blockIdx.x] && ci < ncrow[blockIdx.x] )
                        ( (Float*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ] -= ( (Float*) ( d_C[blockIdx.x] ) )  [ cj * ldc[blockIdx.x] + ci ];
        }
        else
        {
            for ( cj = threadIdx.x; cj < nccol[blockIdx.x]; cj += blockDim.x )
                for ( ci = threadIdx.y; ci < nccol[blockIdx.x]; ci += blockDim.x )
                    if ( cj < nccol[blockIdx.x] && ci < ncrow[blockIdx.x] )
                    {
                        ( (Complex*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ].x -= ( (Complex*) ( d_C[blockIdx.x] ) ) [ cj * ldc[blockIdx.x] + ci ].x;
                        ( (Complex*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ].y -= ( (Complex*) ( d_C[blockIdx.x] ) ) [ cj * ldc[blockIdx.x] + ci ].y;
                    }
        }
    }
    else
    {
        if ( !isComplex )
        {
            for ( cj = threadIdx.x; cj < nccol[blockIdx.x]; cj += blockDim.x )
                for ( ci = threadIdx.y; ci < nccol[blockIdx.x]; ci += blockDim.x )
                    if ( cj < nccol[blockIdx.x] && ci < ncrow[blockIdx.x] )
                        atomicAdd ( & ( ( (Float*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ] ), - ( (Float*) ( d_C[blockIdx.x] ) ) [ cj * ldc[blockIdx.x] + ci ] );
        }
        else
        {
            for ( cj = threadIdx.x; cj < nccol[blockIdx.x]; cj += blockDim.x )
                for ( ci = threadIdx.y; ci < nccol[blockIdx.x]; ci += blockDim.x )
                    if ( cj < nccol[blockIdx.x] && ci < ncrow[blockIdx.x] )
                    {
                        atomicAdd ( & ( ( (Complex*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ].x ), - ( (Complex*) ( d_C[blockIdx.x] ) ) [ cj * ldc[blockIdx.x] + ci ].x );
                        atomicAdd ( & ( ( (Complex*) ( d_A[blockIdx.x] ) ) [ d_RelativeMap[blockIdx.x][cj] * lda[blockIdx.x] + d_RelativeMap[blockIdx.x][ci] ].y ), - ( (Complex*) ( d_C[blockIdx.x] ) ) [ cj * ldc[blockIdx.x] + ci ].y );
                    }
        }
    }
}

void mappedSubtract_batched ( Long batchSize, int isAtomic, int isComplex, void **d_A, Long *lda, void **d_C, Long *nccol, Long *ncrow, Long *ldc, Long **d_RelativeMap, hipStream_t stream )
{
    dim3 block;
    dim3 thread(16, 16);

    block.x = batchSize;

    mappedSubtract_batched_kernel <<< block, thread, 0, stream >>> ( isAtomic, isComplex, d_A, lda, d_C, nccol, ncrow, ldc, d_RelativeMap );
}

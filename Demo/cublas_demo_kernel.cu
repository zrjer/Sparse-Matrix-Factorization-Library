#include "hip/hip_runtime.h"
#include "cublas_demo.h"
#include "cublas_demo_kernel.cuh"

__global__ void launch_syrk_kernel ( struct syrk_meta *d_syrk_task )
{
    __shared__ struct syrk_meta syrk_task;
    __shared__ double shA[DIM_K][DIM_N+PAD];

    int idx;

    int n, k, lda, ldc;
    double alpha, beta;
    double *A, *C;

    idx = blockIdx.x;

    if ( threadIdx.x == 0 && threadIdx.y == 0 )
        syrk_task = d_syrk_task[idx];

    __syncthreads();

    n = syrk_task.n;
    k = syrk_task.k;
    lda = syrk_task.lda;
    ldc = syrk_task.ldc;
    alpha = syrk_task.alpha;
    beta = syrk_task.beta;
    A = syrk_task.A;
    C = syrk_task.C;

    for ( int j = threadIdx.x; j < k; j += blockDim.x )
        for ( int i = threadIdx.y; i < n; i += blockDim.y )
            shA[j][i] = A [ j * lda + i ];

    __syncthreads();

    for ( int j = threadIdx.x; j < n; j += blockDim.x )
        for ( int i = threadIdx.y; i < n; i += blockDim.y )
        {
            double regC;
            regC = beta * C [ j * ldc + i ];
            for ( int kk = 0; kk < k; kk++ )
                regC += ( alpha * shA[kk][j] * shA[kk][i] );
            C[ j * ldc + i ] = regC;
        }
}

__global__ void launch_gemm_kernel ( struct gemm_meta *d_gemm_task )
{
    __shared__ struct gemm_meta gemm_task;
    __shared__ double shA[DIM_K][DIM_M+PAD], shB[DIM_K][DIM_N+PAD];

    int idx;

    int m, n, k, lda, ldb, ldc;
    double alpha, beta;
    double *A, *B, *C;

    idx = blockIdx.x;

    if ( threadIdx.x == 0 && threadIdx.y == 0 )
        gemm_task = d_gemm_task[idx];

    __syncthreads();

    m = gemm_task.m;
    n = gemm_task.n;
    k = gemm_task.k;
    lda = gemm_task.lda;
    ldb = gemm_task.ldb;
    ldc = gemm_task.ldc;
    alpha = gemm_task.alpha;
    beta = gemm_task.beta;
    A = gemm_task.A;
    B = gemm_task.B;
    C = gemm_task.C;

    for ( int j = threadIdx.y; j < k; j += blockDim.y )
        for ( int i = threadIdx.x; i < m; i += blockDim.x )
            shA[j][i] = A [ j * lda + i ];

    for ( int j = threadIdx.x; j < k; j += blockDim.x )
        for ( int i = threadIdx.y; i < n; i += blockDim.y )
            shB[j][i] = B [ j * ldb + i ];

    __syncthreads();

    for ( int j = threadIdx.x; j < n; j += blockDim.x )
        for ( int i = threadIdx.y; i < m; i += blockDim.y )
        {
            double regC;
            regC = beta * C [ j * ldc + i ];
            for ( int kk = 0; kk < k; kk++ )
                regC += ( alpha * shA [kk][i] * shB [kk][j] );
            C [ j * ldc + i ] = regC;
        }
}

void launch_syrk_gemm ( int batch, struct syrk_meta *d_syrk_task, struct gemm_meta *d_gemm_task, hipStream_t stream )
{
    dim3 thread;

    thread.x = 16;
    thread.y = 64;

    launch_syrk_kernel <<<batch, thread, 0, stream>>> ( d_syrk_task );
    launch_gemm_kernel <<<batch, thread, 0, stream>>> ( d_gemm_task );
}
